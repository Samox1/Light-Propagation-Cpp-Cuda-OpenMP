#include <iostream>
#include <fstream>
#include <stdio.h>
#include <math.h>
#include <complex>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
//#include <omp.h>
//#include <mpi.h>

using namespace std;

__global__ void multiplyElementwise(hipfftDoubleComplex* f0, hipfftDoubleComplex* f1, int size)
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < size)
    {
        double a, b, c, d;
        a = f0[i].x; 
        b = f0[i].y;
        c = f1[i].x; 
        d = f1[i].y;
        f0[i].x = a*c - b*d;
        f0[i].y = a*d + b*c;
    }
}


// --- Main Part --- //

int main(int argc, char *argv[])
{

	cout << "Welcome to CUDA test" << endl;
	
	return 0;
} 

