#include <iostream>
#include <fstream>
#include <stdio.h>
#include <math.h>
#include <complex>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
//#include <omp.h>
//#include <mpi.h>

using namespace std;

__global__ void multiplyElementwise(hipfftDoubleComplex* f0, hipfftDoubleComplex* f1, int size)
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < size)
    {
        double a, b, c, d;
        a = f0[i].x;
        b = f0[i].y;
        c = f1[i].x;
        d = f1[i].y;
        f0[i].x = a*c - b*d;
        f0[i].y = a*d + b*c;
    }
}


// void u_in_in_big(double* u_in, cufftDoubleComplex* data, int NX, int NY, int multi);
// void h_z(double lam, double z, double k, double sampling, int NX, int NY, cufftDoubleComplex* h_z_cutab);
// void Q_roll(cufftDoubleComplex* u_in_fft, cufftDoubleComplex* data, int NX, int NY);
// void amplitude_print(cufftDoubleComplex* u_in_fft, int NX, int NY, FILE* fp);
// int FFT_Z2Z(cufftDoubleComplex* dData, int NX, int NY);
// int IFFT_Z2Z(cufftDoubleComplex* dData, int NX, int NY);



// ----------------------------------------------------------------------------------------------------------------------------------------------- //
// --- Functions --- Functions --- Functions --- Functions --- Functions --- Functions --- Functions --- Functions --- Functions --- Functions --- //
// ----------------------------------------------------------------------------------------------------------------------------------------------- //

// void u_in_in_big(double* u_in, cufftDoubleComplex* data, int NX, int NY, int multi)
// {
// 	for(int ii=0; ii < NY ; ii++)
// 	{
// 		for(int jj=0; jj < NX ; jj++)
// 		{
// 			data[ii*NX+jj].x = 0;
// 			data[ii*NX+jj].y = 0;
// 		}
// 	}

// 	for(int ii=0; ii < (int)NY/multi ; ii++)
// 	{
// 		for(int jj=0; jj < (int)NX/multi ; jj++)
// 		{
// 			data[(ii*NX+jj)+(NX*NY*(multi-1)/(multi*2)+NX*(multi-1)/(multi*2))].x = u_in[ii*(NX/multi)+jj];
// 		}
// 	}
// }


// void hz(double lam, double z, double k, double sampling, int NX, int NY, cufftDoubleComplex* hz_cutab)
// {
// 	std::complex<double>* hz_tab;
// 	hz_tab = (std::complex<double> *) malloc ( sizeof(std::complex<double>)* NX * NY);

// 	double fi = k * z;
// 	double teta = k / (2.0 * z);
// 	double lam_z = lam * z;
// 	double quad = 0.0;
// 	double teta1 = 0.0;	

// 	for(int iy=0; iy < NY; iy++)
// 	{
// 		//printf("\n");
// 		for(int ix=0; ix < NX ; ix++)
// 		{
// 			quad = pow(((double)ix-((double)NX/2.0))*sampling, 2) + pow(((double)iy-((double)NY/2.0))*sampling, 2);
// 			teta1 = teta * quad;
// 			hz_tab[iy*NX+ix] = exp(1i*fi)*exp(1i*teta1)/(1i*lam_z);
// 			hz_cutab[iy*NX+ix].x = hz_tab[iy*NX+ix].real();
// 			hz_cutab[iy*NX+ix].y = hz_tab[iy*NX+ix].imag();
// 			//printf("%.2f\t", hz_cutab[iy*NX+ix].x);
// 		}
// 	}	
// 	free(hz_tab);
// }


// void Qroll(cufftDoubleComplex* u_in_fft, cufftDoubleComplex* data, int NX, int NY)
// {
// 	for(int iy=0; iy<(NY/4); iy++)	//Petla na przepisanie tablicy koncowej
// 	{
// 		for(int jx=0; jx<(NX/4); jx++)
// 		{
// 			u_in_fft[(NX/2*NY/4+NY/4)+(jx+iy*NX/2)] = data[iy*(NX)+jx];		// Q1 -> Q4
// 			u_in_fft[(jx+NX/4)+(iy*NX/2)] = data[(iy*(NX)+jx)+(NX*NY*3/4)];		// Q3 -> Q2
// 			u_in_fft[(jx)+(iy*NX/2)] = data[((iy*NX)+jx)+(NX*3/4+NX*NY*3/4)];	// Q4 -> Q1
// 			u_in_fft[(jx)+(iy*NX/2)+NX*NY/2/4] = data[((iy*NX)+jx)+(NX*3/4)];	// Q2 -> Q3
// 		}
// 	}
// }

// void amplitude_print(cufftDoubleComplex* u_in_fft, int NX, int NY, FILE* fp)
// {
// 	// --- Przeliczanie Amplitudy --- //

// 	for(int ii=0; ii<(NX*NY/4); ii++)
// 	{	
// 		u_in_fft[ii].x = sqrt(pow(u_in_fft[ii].x, 2) + pow(u_in_fft[ii].y, 2));
// 	}
	
// 	double mini_data = u_in_fft[0].x;
	
// 	for(int ii=0; ii<(NX*NY/4); ii++)
// 	{		
// 		if (u_in_fft[ii].x < mini_data){ mini_data = u_in_fft[ii].x; }
// 	}
	
// 	double max_data = u_in_fft[0].x;
// 	mini_data = -mini_data;
	
// 	for(int ii=0; ii<(NX*NY/4); ii++)
// 	{		
// 		u_in_fft[ii].x = u_in_fft[ii].x + mini_data;
// 		if (u_in_fft[ii].x > max_data) { max_data = u_in_fft[ii].x; }
// 	}

// 	for(int ii=0; ii<(NX*NY/4); ii++)
// 	{	
// 		if (ii%(NX/2) == 0){fprintf (fp,"\n");}
// 		u_in_fft[ii].x = u_in_fft[ii].x / max_data * 255.0;
// 		fprintf (fp,"%.0f\t", u_in_fft[ii].x);
// 	}
// }

// int FFT_Z2Z(cufftDoubleComplex* dData, int NX, int NY)
// {
// 	// Create a 2D FFT plan. 
// 	int err = 0;
// 	cufftHandle plan1;
// 	if (cufftPlan2d(&plan1, NX, NY, CUFFT_Z2Z) != CUFFT_SUCCESS){
// 		fprintf(stderr, "CUFFT Error: Unable to create plan\n");
// 		err = -1;	
// 	}

// 	if (cufftExecZ2Z(plan1, dData, dData, CUFFT_FORWARD) != CUFFT_SUCCESS){
// 		fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
// 		err = -1;		
// 	}

// 	if (cudaDeviceSynchronize() != cudaSuccess){
//   		fprintf(stderr, "Cuda error: Failed to synchronize\n");
//    		err = -1;
// 	}	
	
// 	cufftDestroy(plan1);
// 	return err;
// }

// int IFFT_Z2Z(cufftDoubleComplex* dData, int NX, int NY)
// {
// 	// Create a 2D FFT plan.
// 	int err = 0; 
// 	cufftHandle plan1;
// 	if (cufftPlan2d(&plan1, NX, NY, CUFFT_Z2Z) != CUFFT_SUCCESS){
// 		fprintf(stderr, "CUFFT Error: Unable to create plan\n");
// 		err = -1;	
// 	}

// 	if (cufftExecZ2Z(plan1, dData, dData, CUFFT_INVERSE) != CUFFT_SUCCESS){
// 		fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
// 		err = -1;		
// 	}

// 	if (cudaDeviceSynchronize() != cudaSuccess){
//   		fprintf(stderr, "Cuda error: Failed to synchronize\n");
//    		err = -1;
// 	}

// 	cufftDestroy(plan1);	
// 	return err;
// }


// --- Main Part --- //

int main(int argc, char *argv[])
{

    cout << "Welcome to CUDA test" << endl;

    int COL = atoi(argv[2]);
	int ROW = atoi(argv[3]);
	//int COL = 1024;
	//int ROW = 1024;
	//double u_in[ROW*COL];
	//cout << "DEBUG" << endl;
	double* u_in;
	u_in = (double *) malloc ( sizeof(double)* COL * ROW);


	//cout << "DUPA WELCOME" << " | " << argv[0] << " | " << argv[1] << " | " << endl;
	cout << "DUPA WELCOME" << " | " << argv[0] << " | " << argv[1] << " | " << argv[2] << " | " << argv[3] << " | " << atoi(argv[4]) << endl;
	//cout << "ROW: " << ROW << " | " << "COL: " << COL <<endl;


	ifstream inputFile;
    inputFile.open(argv[1]);

    if (inputFile)
	{
		cout << "Import file: " << argv[1] << endl;
		int i,j = 0;
		for (i = 0; i < ROW; i++)
		{
			for (j = 0; j < COL; j++)
			{
				inputFile >> u_in[i*ROW+j];
			}
		}
		cout << "Import file - complete" << endl;
	} else {
		cout << "Error opening the file.\n";
	}
	inputFile.close();



	int multi = atoi(argv[4]);
	int NX = COL*multi;
	int NY = ROW*multi;

// --- Przeliczenie hz --- //

	double sampling = 10.0 * pow(10.0, (-6)); 	// Sampling = 10 micro
	double lam = 633.0 * (pow(10.0,(-9))); 		// Lambda = 633 nm
	double k = 2.0 * M_PI / lam;				// Wektor falowy k
	double z_in = 500.0*(pow(10.0,(-3)));		// Odleglosc propagacji = 0,5 m
	double z_out = 1000.0*(pow(10.0,(-3)));     // Koniec odległości propagacji = 1 m
	double z_delta = 50.0*(pow(10.0,(-3)));     // Skok odległości = 0,05 m
	//double z = z_in+(ip*z_delta);             // Odległość Z dla każdego wątku MPI
    double z = z_in;

    printf("k = %.1f | lam = %.1f nm | z = %.4f m | \n", k, lam*(pow(10.0,(9))), z);

// 	// --- FFT tablicy wejsciowej --- //
// 	cufftDoubleComplex* data;
// 	data = (cufftDoubleComplex *) malloc ( sizeof(cufftDoubleComplex)* NX * NY);

// 	cufftDoubleComplex* dData;
// 	cudaMalloc((void **) &dData, sizeof(cufftDoubleComplex)* NX * NY);

// 	if (cudaGetLastError() != cudaSuccess){
// 		fprintf(stderr, "Cuda error: Failed to allocate\n");
// 		return -1;
// 	}
	
// 	size_t pitch1;

// 	u_in_in_big(u_in, data, NX, NY, multi);

// 	// Liczenie U_in = FFT{u_in}
//  	cudaMallocPitch(&dData, &pitch1, sizeof(cufftDoubleComplex)*NX, NY);
// 	cudaMemcpy2D(dData,pitch1,data,sizeof(cufftDoubleComplex)*NX,sizeof(cufftDoubleComplex)*NX,NX,cudaMemcpyHostToDevice);
 	
// 	if (cudaGetLastError() != cudaSuccess){
// 		fprintf(stderr, "Cuda error: Failed to allocate\n");
// 		return -1;	
// 	}

// 	if (FFT_Z2Z(dData, NX, NY) == -1) { return -1; }
// 		cudaMemcpy(data, dData, sizeof(cufftDoubleComplex)*NX*NY, cudaMemcpyDeviceToHost);
// 	}	
	
// // Liczenie hz

// 	cufftDoubleComplex* hz_tab;
// 	hz_tab = (cufftDoubleComplex *) malloc ( sizeof(cufftDoubleComplex)* NX * NY);
// 	hz(lam, z, k, sampling, NX, NY, hz_tab);	

// // --- Liczenie hz = FFT{hz_tab} --- //
	
// 	cufftDoubleComplex* hz;
// 	cudaMalloc((void **) &hz, sizeof(cufftDoubleComplex)* NX * NY);

// 	size_t pitch2;
//  	cudaMallocPitch(&hz, &pitch2, sizeof(cufftDoubleComplex)*NX, NY);
// 	cudaMemcpy2D(hz,pitch2,hz_tab,sizeof(cufftDoubleComplex)*NX,sizeof(cufftDoubleComplex)*NX,NX,cudaMemcpyHostToDevice);

// 	if(cudaGetLastError() != cudaSuccess){
// 		fprintf(stderr, "Cuda error: Failed to allocate\n");
// 		return -1;	
// 	}

// 	if (FFT_Z2Z(hz, NX, NY) == -1) { return -1; }

// 	// Do the actual multiplication

// 	multiplyElementwise<<<NX*NY, 1>>>(dData, hz, NX*NY);
	

// // --- Liczenie u_out = iFFT{dData = U_OUT} --- //

// 	if(IFFT_Z2Z(dData, NX, NY) == -1) { return -1; }

// 	cudaMemcpy(data, dData, sizeof(cufftDoubleComplex)*NX*NY, cudaMemcpyDeviceToHost);

// 	//printf( "\nCUFFT vals: \n");
	
// // Czytanie calosci


// // --- ROLL cwiartek, zeby wszystko sie zgadzalo na koniec --- //

// 	cufftDoubleComplex* u_out;
// 	u_out = (cufftDoubleComplex *) malloc (sizeof(cufftDoubleComplex)* NX/2 * NY/2);

// 	Qroll(u_out, data, NX, NY);

// // --- Przeliczanie Amplitudy --- //

// 	char filename[128];
// 	snprintf ( filename, 128, "result_z_%.5lf.txt", z );
// 	FILE* fp = fopen(filename,"w");

// 	amplitude_print(u_out, NX, NY, fp);

// 	fclose(fp);

// 	cudaFree(u_out);
// 	cudaFree(data);
// 	cudaFree(dData);
// 	cudaFree(hz_tab);
// 	cudaFree(hz);

	free(u_in);

	return 0;
}


